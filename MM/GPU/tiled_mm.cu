#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_WIDTH 32

__global__ void tiled_mm_kernel(int32_t *a, int32_t *b, int32_t *c, int M, int K, int N) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * BLOCK_WIDTH + ty;
    int Col = bx * BLOCK_WIDTH + tx;

    int32_t value = 0;

    __shared__ int32_t A_block[BLOCK_WIDTH][BLOCK_WIDTH];
    __shared__ int32_t B_block[BLOCK_WIDTH][BLOCK_WIDTH];

    for (int index = 0; index < ceil(K / (float)BLOCK_WIDTH); index++) {
        if ((Row < M) && (index * BLOCK_WIDTH + tx < K))
            A_block[ty][tx] = a[Row * K + index * BLOCK_WIDTH + tx];
        else
            A_block[ty][tx] = 0;

        if ((Col < N) && (index * BLOCK_WIDTH + ty) < K)
            B_block[ty][tx] = b[(index * BLOCK_WIDTH + ty) * N + Col];
        else
            B_block[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < BLOCK_WIDTH; k++) {
            value += A_block[ty][k] * B_block[k][tx];
        }

        __syncthreads();

        if ((Row < M) && (Col < N))
            c[Row * N + Col] = value;
    }
}

int Test_mm(int32_t *A, int32_t *B, int32_t *C, int M, int K, int N) {
    printf("Verifying all results...\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            int32_t val = 0;
            for (int k = 0; k < K; k++) {
                val += A[i * K + k] * B[k * N + j];
            }

            if (val - C[i * N + j] != 0) {
                printf("%d != %d\n", C[i * N + j], val);
                fprintf(stderr, "Result verification failed at (%d, %d)\n", i, j);
                fprintf(stderr, "Test FAILED\n");
                return -1;
            }

            // printf("%5d", val);
        }
        // printf("\n");
    }
    printf(".....\n");
    printf("Test PASSED\n");

    return 0;
}

int Just_mm(int32_t *A, int32_t *B, int32_t *C, int M, int K, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            int32_t val = 0;
            for (int k = 0; k < K; k++) {
                val += A[i * K + k] * B[k * N + j];
            }
        }
    }
    return 0;
}

int print_matrix(int32_t *C, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%5d", C[i * N + j]);
        }
        printf("\n");
    }
    printf(".....\n");

    return 0;
}

int main(void) {
    int32_t *a, *b, *c;  // memory in host
    int *d_a, *d_b, *d_c;
    int M, N, K;
    int size = sizeof(int32_t);

    M = 1024;
    N = 1024;
    K = 1024;

    a = (int32_t *)malloc(M * K * size);
    b = (int32_t *)malloc(K * N * size);
    c = (int32_t *)malloc(M * N * size);

    // initialize matrix
    for (int i = 0; i < M * K; i++) {
        a[i] = i;
    }
    for (int i = 0; i < K * N; i++) {
        b[i] = i;
    }

    hipMalloc((void **)&d_a, M * K * size);
    hipMalloc((void **)&d_b, K * N * size);
    hipMalloc((void **)&d_c, M * N * size);

    dim3 threadsPerBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 blockPerGrid(ceil(M / (float)BLOCK_WIDTH), ceil(N / (float)BLOCK_WIDTH));

    int start = clock();

    hipMemcpy(d_a, a, M * K * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, K * N * size, hipMemcpyHostToDevice);

    tiled_mm_kernel<<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, M, K, N);

    hipMemcpy(c, d_c, M * N * size, hipMemcpyDeviceToHost);

    int end = clock();
    int tiling_time = end - start;

    start = clock();
    Just_mm(a, b, c, M, K, N);
    end = clock();

    int just_mm_time = end - start;
    Test_mm(a, b, c, M, K, N);

    printf("Compare Time\n");
    printf("ijk mm time : %d\n", just_mm_time);
    printf("Tiling mm time : %d\n", tiling_time);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
